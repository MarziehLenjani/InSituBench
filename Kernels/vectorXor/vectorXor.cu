#include "hip/hip_runtime.h"
/**
 * Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

/**
 * Vector addition: C = A + B.
 *
 * This sample is a very basic sample that implements element by element
 * vector addition. It is the same as the sample illustrating Chapter 2
 * of the programming guide with some additions like error checking.
 */

#include <stdio.h>

// For the CUDA runtime routines (prefixed with "cuda_")
#include <hip/hip_runtime.h>
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>
#include <VaryTypeAndOperator.h>
#include <PrintOutput.h>
/**
 * CUDA Kernel Device code
 *
 * Computes the vector addition of A and B into C. The 3 vectors have the same
 * number of elements numElements.
 */
__global__ void
vectorAdd(const int *A, const int *B, int *C, int numElements)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < numElements)
    {
        C[i] = A[i] ^ B[i];
    }
}

/**
 * Host main routine
 */
int
main(int argc, char * argv[])
{
    // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;
    int numElements = 50000;
    if (checkCmdLineFlag(argc, (const char **)argv, "nIter"))
    {
    		nIter = getCmdLineArgumentInt(argc, (const char **)argv, "nIter");
    }else{
		#ifndef METRIC_RUN_ONLY_ONCE
			nIter = 30;
		#else
			nIter = 1;
		#endif
	}
    if (checkCmdLineFlag(argc, (const char **)argv, "Num"))
     {
    		numElements = getCmdLineArgumentInt(argc, (const char **)argv, "Num");
     }
    // Print the vector length to be used, and compute its size

    size_t size = numElements * sizeof(int);
    printf("[Vector addition of %d elements]\n", numElements);
    noWarmUp=checkCmdLineFlag(argc, (const char **)argv, "NoWarmUp");


    // Allocate the host input vector A
    int *h_A = (int *)malloc(size);

    // Allocate the host input vector B
    int *h_B = (int *)malloc(size);

    // Allocate the host output vector C
    int *h_C = (int *)malloc(size);

    // Verify that allocations succeeded
    if (h_A == NULL || h_B == NULL || h_C == NULL)
    {
        fprintf(stderr, "Failed to allocate host vectors!\n");
        exit(EXIT_FAILURE);
    }

    // Initialize the host input vectors
    for (int i = 0; i < numElements; ++i)
    {
        h_A[i] = rand()/(int)RAND_MAX;
        h_B[i] = rand()/(int)RAND_MAX;
    }

    // Allocate the device input vector A
    int *d_A = NULL;
    err = hipMalloc((void **)&d_A, size);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector A (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Allocate the device input vector B
    int *d_B = NULL;
    err = hipMalloc((void **)&d_B, size);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector B (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Allocate the device output vector C
    int *d_C = NULL;
    err = hipMalloc((void **)&d_C, size);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector C (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Copy the host input vectors A and B in host memory to the device input vectors in
    // device memory
    printf("Copy input data from the host memory to the CUDA device\n");
    err = hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector A from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector B from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Launch the Vector Add CUDA Kernel
    int threadsPerBlock = 256;
    int blocksPerGrid =(numElements + threadsPerBlock - 1) / threadsPerBlock;
    printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);
    //--------------profiling
    float msecTotal = 0.0f;
    hipEvent_t start, stop;
	checkCudaErrors(hipEventCreate(&start));
	checkCudaErrors(hipEventCreate(&stop));
	// Record the start event
	checkCudaErrors(hipEventRecord(start, NULL));
    for(int t=0; t< nIter; t++)
    {
    		vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, numElements);
    }
	checkCudaErrors(hipEventRecord(stop, NULL));

	// Wait for the stop event to complete
	checkCudaErrors(hipEventSynchronize(stop));

	checkCudaErrors(hipEventElapsedTime(&msecTotal, start, stop));
    err = hipGetLastError();
    double msec = msecTotal / nIter;
     sizeInGBytes= (sizeof(int)*numElements*2)* 1.0e-9;
     if(msec!=0){
     	gigaProcessedInSec=( sizeInGBytes / (msec / 1000.0f));
     }
     outPutSizeInGBytes=sizeof(int)*numElements*1.0e-9;
     timeInMsec=msec;
     printOutput();
     printf("nIter %d\n", nIter);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to launch vectorAdd kernel (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Copy the device result vector in device memory to the host result vectorAdd
    // in host memory.
    printf("Copy output data from the CUDA device to the host memory\n");
    err = hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector C from device to host (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Verify that the result vector is correct
    for (int i = 0; i < numElements; ++i)
    {
        if (fabs(h_A[i] + h_B[i] - h_C[i]) > 1e-5)
        {
            fprintf(stderr, "Result verification failed at element %d!\n", i);
            exit(EXIT_FAILURE);
        }
    }

    printf("Test PASSED\n");

    // Free device global memory
    err = hipFree(d_A);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector A (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipFree(d_B);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector B (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipFree(d_C);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector C (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Free host memory
    free(h_A);
    free(h_B);
    free(h_C);

    // Reset the device and exit
    // hipDeviceReset causes the driver to clean up all state. While
    // not mandatory in normal operation, it is good practice.  It is also
    // needed to ensure correct operation when the application is being
    // profiled. Calling hipDeviceReset causes all profile data to be
    // flushed before the application exits
    err = hipDeviceReset();

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to deinitialize the device! error=%s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    printf("Done\n");
    return 0;
}

